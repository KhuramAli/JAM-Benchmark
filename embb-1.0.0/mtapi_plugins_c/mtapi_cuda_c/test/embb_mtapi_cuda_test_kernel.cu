
#include <hip/hip_runtime.h>
extern "C" __global__ void test(
  void* arguments,
  int arguments_size,
  void* result_buffer,
  int result_buffer_size,
  void* node_local_data,
  int node_local_data_size) {
  int ii = blockDim.x * blockIdx.x + threadIdx.x;
  int elements = arguments_size / sizeof(float) / 2;
  if (ii >= elements)
    return;
  float* a = (float*)arguments;
  float* b = ((float*)arguments) + elements;
  float* c = (float*)result_buffer;
  float* d = (float*)node_local_data;
  c[ii] = a[ii] + b[ii] + d[0];
}
